

#include <hip/hip_runtime.h>
#include <stdio.h> 
#include <stdlib.h> 
#include <math.h> 
#include <assert.h> 
#include <time.h> 

#include <sys/time.h>

#include <hiprand/hiprand_kernel.h> 

#define D 5

#define TRIALS_PER_THREAD 2048
#define BLOCKS 256
#define THREADS 256 

__global__ void mc_int(double *res, hiprandState *states) { 
	unsigned int tid = threadIdx.x + blockDim.x*blockIdx.x;
	double integral = 0.0; 
	double X[D]; 

	hiprand_init(tid, 0, 0, &states[tid]); 
	for (int i = 0; i < TRIALS_PER_THREAD; i++) { 
		for (int j = 0; j < D; j++) { 
			X[j] = hiprand_uniform(&states[tid]); 
		}
		double t = 0.0; 
		for (int j = 0; j < D; j++) { 
			t -= X[j] * X[j]; 
		}
		integral += exp(t) / TRIALS_PER_THREAD; 
	}
	res[tid] = integral; 
}

int main(int argc, char **argv) { 

	double host[BLOCKS * THREADS]; 
	double *dev; 
	hiprandState *states; 

	double integral = 0.0; 
	double vol = 1.0; 

	clock_t ts = clock(); 
	struct timeval start, end;
	gettimeofday(&start, NULL);

	hipMalloc((void**) &dev, BLOCKS * THREADS * sizeof(double));
	hipMalloc((void**)&states, 
			BLOCKS * THREADS * sizeof(hiprandState));

	mc_int<<<BLOCKS, THREADS>>>(dev, states); 	

	hipMemcpy(host, dev, BLOCKS * THREADS * sizeof(double), 
			hipMemcpyDeviceToHost); 

	for(int i = 0; i < BLOCKS * THREADS; i++) {
		integral += host[i];
	}
	integral /= BLOCKS * THREADS;

	for (int j = 0; j < D; j++) { 
		vol *= 1.0; 
	}
	integral *= vol; 

	gettimeofday(&end, NULL);
	double elapsed = ((end.tv_sec  - start.tv_sec) * 1000000u + 
			         end.tv_usec - start.tv_usec) / 1.e6;
	ts = clock() - ts; 

	printf("%ld clocks (%lf seconds)\n", ts, elapsed); 
	printf("integral is: %lf\n", integral);	
	hipFree(dev);
	hipFree(states);
}
