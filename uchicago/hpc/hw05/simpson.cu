

#include <hip/hip_runtime.h>
#include <stdio.h> 
#include <stdlib.h> 
#include <math.h> 
#include <assert.h> 
#include <time.h> 

#include <sys/time.h>

#include <hiprand/hiprand_kernel.h> 

#define D 5

#define BLOCKS 125
#define THREADS 25
#define N 5

__global__ void simpson_int(double *res) { 
	unsigned int tid = threadIdx.x + blockDim.x*blockIdx.x;
	int t = tid; 
	double integral = 0.0; 
	double X[D]; 

	X[0] = t % N; t /= N; 
	X[1] = t % N; t /= N; 
	X[2] = t % N; t /= N; 
	X[3] = t % N; t /= N; 
	X[4] = t % N; 
	
	double T = 0.0; 
	for (int j = 0; j < D; j++) { 
		X[j] = X[j] / N; 
		T -= X[j] * X[j]; 
	}
	integral += exp(T) * pow(1.0/N, 5.0);
	res[tid] = integral; 
}

int main(int argc, char **argv) { 

	double host[BLOCKS * THREADS]; 
	double *dev; 

	double integral = 0.0; 
	double vol = 1.0; 

	clock_t ts = clock(); 
	struct timeval start, end;
	gettimeofday(&start, NULL);

	hipMalloc((void**) &dev, BLOCKS * THREADS * sizeof(double));

	simpson_int<<<BLOCKS, THREADS>>>(dev); 	

	hipMemcpy(host, dev, BLOCKS * THREADS * sizeof(double), 
			hipMemcpyDeviceToHost); 

	for(int i = 0; i < BLOCKS * THREADS; i++) {
		integral += host[i];
	}

	for (int j = 0; j < D; j++) { 
		vol *= 1.0; 
	}
	integral *= vol; 

	gettimeofday(&end, NULL);
	double elapsed = ((end.tv_sec  - start.tv_sec) * 1000000u + 
			         end.tv_usec - start.tv_usec) / 1.e6;
	ts = clock() - ts; 

	printf("%ld clocks (%lf seconds)\n", ts, elapsed); 
	printf("integral is: %lf\n", integral);	
	hipFree(dev);
}
